
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <string>
#include <fstream>

using std::ifstream;
using std::ofstream;
using std::cout;
using std::cin;
using std::cerr;
using std::endl;
using std::string;
using std::ios_base;

#define CSC(call)                                                               \
    {                                                                           \
        auto error = call;                                                      \
        if (error != hipSuccess) {                                             \
            cerr << "Error " << hipGetErrorName(error) << " in file \""        \
                 << __FILE__ << "\", at line " << __LINE__ << ". "              \
                 << "Message: " << hipGetErrorString(error) << endl;           \
            exit(1);                                                            \
        }                                                                       \
    }

struct size2D {
    int width, height;

    int getSize() const {
        return width * height;
    }
};

__global__ void kernel(hipTextureObject_t source, uchar4 *target,
                       size2D sourceSize, size2D targetSize) {
    auto totalThreadsCountX = gridDim.x * blockDim.x;
    auto totalThreadsCountY = gridDim.y * blockDim.y;
    auto threadIdX = blockDim.x * blockIdx.x + threadIdx.x;
    auto threadIdY = blockDim.y * blockIdx.y + threadIdx.y;

    auto xCompressionCoefficient = sourceSize.width / targetSize.width;
    auto yCompressionCoefficient = sourceSize.height / targetSize.height;
    auto compressionBlockSize = xCompressionCoefficient * yCompressionCoefficient;

    for (unsigned int i = threadIdX; i < targetSize.width; i += totalThreadsCountX) {
        for (unsigned int j = threadIdY; j < targetSize.height; j += totalThreadsCountY) {

            auto rSum = 0, gSum = 0, bSum = 0, aSum = 0;
            for (int xShift = 0; xShift < xCompressionCoefficient; ++xShift) {
                for (int yShift = 0; yShift < yCompressionCoefficient; ++yShift) {
                    auto x = xCompressionCoefficient * i + xShift;
                    auto y = yCompressionCoefficient * j + yShift;
                    auto pixel = tex2D<uchar4>(source,(float) x,(float) y);
                    rSum += pixel.x;
                    gSum += pixel.y;
                    bSum += pixel.z;
                    aSum += pixel.w;
                }
            }

            target[j * targetSize.width + i] = make_uchar4(
                    rSum / compressionBlockSize,
                    gSum / compressionBlockSize,
                    bSum / compressionBlockSize,
                    aSum / compressionBlockSize);
        }
    }
}

bool isValidSizes(const size2D &sourceSize, const size2D &targetSize) {
    if (targetSize.width > sourceSize.width
        || targetSize.height > sourceSize.height
        || sourceSize.width % targetSize.width != 0
        || sourceSize.height % targetSize.height != 0) {
        return false;
    }

    return true;
}

int main(int argc, char *argv[]) {
    size2D targetSize{};
    string sourceFilePath;
    string targetFilePath;
    if (argc == 5) {
        sourceFilePath = argv[1];
        targetFilePath = argv[2];
        targetSize.width = std::stoi(argv[3]);
        targetSize.height = std::stoi(argv[4]);
    } else {
        cin >> sourceFilePath;
        cin >> targetFilePath;
        cin >> targetSize.width >> targetSize.height;
    }

    // read source
    ifstream input(sourceFilePath, ios_base::binary);
    if (!input.is_open()) {
        cerr << "Could not open source file." << endl;
        return 1;
    }

    size2D sourceSize{};
    input.read((char *) &sourceSize.width, sizeof(sourceSize.width));
    input.read((char *) &sourceSize.height, sizeof(sourceSize.height));

    auto *data = new uchar4[sourceSize.getSize()];
    input.read((char *) data, (long long) sizeof(data[0]) * sourceSize.getSize());
    input.close();

    // validate inputs
    if (!isValidSizes(sourceSize, targetSize)) {
        cerr << "Error: wrong target image size." << endl;
        return 1;
    }

    // move data to device
    hipArray *cudaData;
    auto channel = hipCreateChannelDesc<uchar4>();
    auto sourcePitch = sizeof(uchar4) * sourceSize.width;
    CSC(hipMallocArray(&cudaData, &channel, sourceSize.width, sourceSize.height))
    CSC(hipMemcpy2DToArray(cudaData, 0, 0, data,
                            sourcePitch,
                            sizeof(uchar4) * sourceSize.width,
                            sourceSize.height,
                            hipMemcpyHostToDevice))
    delete[] data;

    // creating texture
    hipTextureObject_t tex;
    hipResourceDesc resDesc{};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cudaData;
    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = false;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr))

    // allocating memory for result
    uchar4 *cudaResult;
    CSC(hipMalloc(&cudaResult, sizeof(uchar4) * targetSize.getSize()))

    // core
    kernel<<<dim3(16, 16), dim3(32, 32)>>>(tex, cudaResult, sourceSize, targetSize);
    CSC(hipDeviceSynchronize())
    CSC(hipGetLastError())

    // move result from device to host
    auto *localResult = new uchar4[targetSize.getSize()];
    CSC(hipMemcpy(localResult, cudaResult, sizeof(localResult[0]) * targetSize.getSize(),
                   hipMemcpyDeviceToHost))
    CSC(hipDestroyTextureObject(tex))
    CSC(hipFreeArray(cudaData))
    CSC(hipFree(cudaResult))

    // save result
    ofstream output(targetFilePath, ios_base::binary);
    if (!output.is_open()) {
        cerr << "Could not open target file." << endl;
        return 1;
    }
    output.write((char *) &targetSize.width, sizeof(targetSize.width));
    output.write((char *) &targetSize.height, sizeof(targetSize.height));
    output.write((char *) localResult, (long long) sizeof(localResult[0]) * targetSize.getSize());
    output.close();

    delete[] localResult;

    return 0;
}
