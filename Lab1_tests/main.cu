
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <vector>
#include <thread>

using std::cin;
using std::cout;
using std::endl;
using std::setprecision;
using std::scientific;
using std::string;
using std::rand;
using std::vector;
using std::thread;

using std::chrono::steady_clock;
using std::chrono::milliseconds;
using std::chrono::duration_cast;

__global__ void kernel(const double *first, const double *second, unsigned int n, double *results) {
    auto totalThreadsCount = gridDim.x * blockDim.x;
    auto currentThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned int i = currentThreadId; i < n; i += totalThreadsCount) {
        results[i] = first[i] > second[i]
                     ? first[i]
                     : second[i];
    }
}

void kernelCpu(const double *first, const double *second, unsigned int n, double *results, int threadsCount,
               int threadIndex) {
    for (int i = threadIndex; i < n; i += threadsCount) {
        results[i] = first[i] > second[i]
                     ? first[i]
                     : second[i];
    }
}

void calcSingleThread(const double *first, const double *second, unsigned int n, double *results) {
    for (int i = 0; i < n; ++i) {
        results[i] = first[i] > second[i]
                     ? first[i]
                     : second[i];
    }
}

void calcMultiThread(const double *first, const double *second, unsigned int n, double *results) {
    auto threadsCount = 10;
    vector<std::thread> threads;

    for (int i = 0; i < threadsCount; i++) {
        threads.emplace_back(kernelCpu, first, second, n, results, threadsCount, i);
    }

    for (auto &th: threads) {
        th.join();
    }
}

long long testCpuMultiThread(int n) {
    cout << "Start cpu test" << endl;

    auto *first = new double[n], *second = new double[n], *results = new double[n];
    for (int i = 0; i < n; ++i) {
        first[i] = rand();
        second[i] = rand();
    }
    cout << "Values init done" << endl;

    auto start = steady_clock::now();
    calcMultiThread(first, second, n, results);
    auto end = steady_clock::now();
    auto elapsedMs = duration_cast<milliseconds>(end - start).count();
    cout << "Multi thread calc done in " << elapsedMs << "ms" << endl;

    cout << "---Testing---" << endl;
    for (int i = 0; i < n; ++i) {
        if (results[i] < first[i] || results[i] < second[i]) {
            cout << "failed: " << first[i] << ", " << second[i] << " -> " << results[i] << endl;
        }
    }
    cout << "---Testing done---" << endl;

    return elapsedMs;
}

long long testCpuSingleThread(int n) {
    cout << "Start cpu test" << endl;

    auto *first = new double[n], *second = new double[n], *results = new double[n];
    for (int i = 0; i < n; ++i) {
        first[i] = rand();
        second[i] = rand();
    }
    cout << "Values init done" << endl;

    auto start = steady_clock::now();
    calcSingleThread(first, second, n, results);
    auto end = steady_clock::now();
    auto elapsedMs = duration_cast<milliseconds>(end - start).count();
    cout << "Single thread calc done in " << elapsedMs << "ms" << endl;

    cout << "---Testing---" << endl;
    for (int i = 0; i < n; ++i) {
        if (results[i] < first[i] || results[i] < second[i]) {
            cout << "failed: " << first[i] << ", " << second[i] << " -> " << results[i] << endl;
        }
    }
    cout << "---Testing done---" << endl;

    return elapsedMs;
}

float testGpu(int n) {

    cout << "Start cuda test" << endl;

    auto *first = new double[n], *second = new double[n];
    for (int i = 0; i < n; ++i) {
        first[i] = rand();
        second[i] = rand();
    }
    cout << "Values init done" << endl;

    double *cudaFirst;
    double *cudaSecond;
    double *cudaResults;
    hipMalloc(&cudaFirst, sizeof(double) * n);
    hipMalloc(&cudaSecond, sizeof(double) * n);
    hipMalloc(&cudaResults, sizeof(double) * n);
    hipMemcpy(cudaFirst, first, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(cudaSecond, second, sizeof(double) * n, hipMemcpyHostToDevice);
    cout << "Cuda values init done" << endl;

    hipEvent_t startEvent, endEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&endEvent);
    hipEventRecord(startEvent);
    kernel<<<1024, 1024>>>(cudaFirst, cudaSecond, n, cudaResults);
    hipEventRecord(endEvent);
    hipEventSynchronize(endEvent);
    float elapsedMs;
    hipEventElapsedTime(&elapsedMs, startEvent, endEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(endEvent);
    cout << "Calc done in " << elapsedMs << "ms" << endl;

    hipFree(cudaFirst);
    hipFree(cudaSecond);

    auto *results = new double[n];
    hipMemcpy(results, cudaResults, sizeof(double) * n, hipMemcpyDeviceToHost);
    hipFree(cudaResults);
    cout << "Results extracted" << endl;

    cout << "---Testing---" << endl;
    for (int i = 0; i < n; ++i) {
        if (results[i] < first[i] || results[i] < second[i]) {
            cout << "failed: " << first[i] << ", " << second[i] << " -> " << results[i] << endl;
        }
    }
    cout << "---Testing done---" << endl;

    delete[] first;
    delete[] second;
    delete[] results;

    return elapsedMs;
}

int main() {
    const int n = 1000000;

    cout << "Values count: " << n << endl << endl;

    auto testCount = 10;

    auto elapsedSum = 0.0;
    for (int i = 0; i < testCount; ++i) {
        cout << "---Test:" << i << "---" << endl;
        elapsedSum += testGpu(n);
        cout << endl;
    }
    cout << "Average elapsed time: " << elapsedSum / testCount << "ms" << endl;


    auto elapsedSingleThreadSum = 0.0;
    auto elapsedMultiThreadSum = 0.0;
    for (int i = 0; i < testCount; ++i) {
        cout << "---Test:" << i << "---" << endl;
        elapsedSingleThreadSum += (double) testCpuSingleThread(n);
        elapsedMultiThreadSum += (double) testCpuMultiThread(n);
        cout << endl;
    }
    cout << "Average elapsed time in single: " << elapsedSingleThreadSum / testCount << "ms" << endl;
    cout << "Average elapsed time in multi: " << elapsedMultiThreadSum / testCount << "ms" << endl;

    return 0;
}
