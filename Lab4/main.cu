#include "hip/hip_runtime.h"
#include <iostream>
#include <tuple>
#include <string>
#include <fstream>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <iomanip>


using std::cout;
using std::cerr;
using std::cin;
using std::endl;
using std::istream;
using std::ostream;
using std::ifstream;
using std::tuple;
using std::string;
using std::setprecision;
using std::scientific;


struct floatAbsComparator {
    __host__ __device__ bool operator()(float a, float b) {
        return abs(a) < abs(b);
    }
};


tuple<int, float *> readAndPrepareInput(istream &input) {
    int matrixSize;
    input >> matrixSize;

    auto *matrix = new float[matrixSize * matrixSize * 2];

    for (auto row = 0; row < matrixSize; ++row) {
        for (auto column = 0; column < matrixSize; ++column) {
            input >> matrix[column * matrixSize + row];
        }
    }

    auto *attachedMatrix = matrix + matrixSize * matrixSize;
    for (auto row = 0; row < matrixSize; ++row) {
        for (auto column = 0; column < matrixSize; ++column) {
            attachedMatrix[column * matrixSize + row] = row == column ? 1 : 0;
        }
    }

    return tuple<int, float *>{matrixSize, matrix};
}


tuple<int, float *> readAndPrepareInput(int argc, char *argv[]) {
    if (argc == 2) {
        auto inputFilePath = argv[1];

        ifstream input(inputFilePath);
        if (!input.is_open()) {
            cerr << "Could not open input file \"" << inputFilePath << "\"." << endl;
            exit(1);
        }

        auto result = readAndPrepareInput(input);

        input.close();

        return result;
    } else {
        return readAndPrepareInput(cin);
    }
}


__global__ void swapRowsKernel(float *matrix, int matrixSize, int i, int j, int startColumn) {
    auto threadCount = gridDim.x * blockDim.x;
    auto threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    for (auto column = startColumn + threadIndex; column < matrixSize * 2; column += threadCount) {
        auto tm = matrix[column * matrixSize + i];
        matrix[column * matrixSize + i] = matrix[column * matrixSize + j];
        matrix[column * matrixSize + j] = tm;
    }
}


__global__ void nullifyRowsBelowKernel(float *matrix, int matrixSize, int diagonalIndex) {
    auto threadCount = gridDim.x * blockDim.x;
    auto threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    for (auto row = diagonalIndex + 1 + threadIndex; row < matrixSize; row += threadCount) {
        auto multiplier = matrix[diagonalIndex * matrixSize + row] / matrix[diagonalIndex * matrixSize + diagonalIndex];
        for (auto column = diagonalIndex + 1; column < matrixSize * 2; ++column) {
            matrix[column * matrixSize + row] -= multiplier * matrix[column * matrixSize + diagonalIndex];
        }
    }
}


__global__ void nullifyRowsAboveKernel(float *matrix, int matrixSize, int diagonalIndex) {
    auto threadCount = gridDim.x * blockDim.x;
    auto threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    for (auto row = threadIndex; row < diagonalIndex; row += threadCount) {
        auto multiplier = matrix[diagonalIndex * matrixSize + row] / matrix[diagonalIndex * matrixSize + diagonalIndex];
        for (auto column = matrixSize; column < matrixSize * 2; ++column) {
            matrix[column * matrixSize + row] -= multiplier * matrix[column * matrixSize + diagonalIndex];
        }
    }
}


__global__ void normalizeDiagonalKernel(float *matrix, int matrixSize) {
    auto threadCount = gridDim.x * blockDim.x;
    auto threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    for (auto diagonalIndex = threadIndex; diagonalIndex < matrixSize; diagonalIndex += threadCount) {
        for (auto column = matrixSize; column < matrixSize * 2; ++column) {
            matrix[column * matrixSize + diagonalIndex] /= matrix[diagonalIndex * matrixSize + diagonalIndex];
        }
    }
}


void writeMatrix(ostream &output, float *matrix, int matrixSize) {
    for (auto row = 0; row < matrixSize; ++row) {
        for (auto column = 0; column < matrixSize; ++column) {
            output << matrix[column * matrixSize + row] << " ";
        }
        output << endl;
    }
}


int main(int argc, char *argv[]) {
    /**
     * Важные нюансы в программе:
     *  Матрица хранится в транспонированном виде, т.е. не по строкам, а по столбцам.
     *  Матрица сразу создается вместе с присоединенной единичной матрицей, т.е. содержит N строк и 2N столбцов.
     */

    int matrixSize;
    float *matrix;
    std::tie(matrixSize, matrix) = readAndPrepareInput(argc, argv);

    float *cudaMatrix;
    hipMalloc(&cudaMatrix, sizeof(float) * matrixSize * matrixSize * 2);
    hipMemcpy(cudaMatrix, matrix, sizeof(float) * matrixSize * matrixSize * 2, hipMemcpyHostToDevice);

    // Forward step
    auto cudaMatrixDevicePtr = thrust::device_pointer_cast(cudaMatrix);
    for (auto diagonalIndex = 0; diagonalIndex < matrixSize; ++diagonalIndex) {
        auto columnPtr = cudaMatrixDevicePtr + diagonalIndex * matrixSize;
        auto maxElementPtr = thrust::max_element(
                columnPtr + diagonalIndex,
                columnPtr + matrixSize,
                floatAbsComparator()
        );
        auto maxElementRowIndex = (int) (maxElementPtr - columnPtr);
        float tm;
        hipMemcpy(
                &tm,
                cudaMatrix + diagonalIndex * matrixSize + maxElementRowIndex,
                sizeof(float),
                hipMemcpyDeviceToHost
        );
        if (tm == 0) {
            cerr << "Could not calculate inverse matrix. Determinant of matrix equal zero." << endl;
            return 1;
        }

        if (diagonalIndex != maxElementRowIndex) {
            swapRowsKernel<<<1024, 1024>>>(cudaMatrix, matrixSize, diagonalIndex, maxElementRowIndex, diagonalIndex);
        }

        nullifyRowsBelowKernel<<<1024, 1024>>>(cudaMatrix, matrixSize, diagonalIndex);
    }

    // Back step
    for (auto diagonalIndex = matrixSize - 1; diagonalIndex > -1; --diagonalIndex) {
        nullifyRowsAboveKernel<<<1024, 1024>>>(cudaMatrix, matrixSize, diagonalIndex);
    }

    // Last step
    normalizeDiagonalKernel<<<1024, 1024>>>(cudaMatrix, matrixSize);

    hipMemcpy(matrix, cudaMatrix, sizeof(float) * matrixSize * matrixSize * 2, hipMemcpyDeviceToHost);
    hipFree(cudaMatrix);

    cout << setprecision(10) << scientific;
    writeMatrix(cout, matrix + matrixSize * matrixSize, matrixSize);

    delete[] matrix;

    return 0;
}
