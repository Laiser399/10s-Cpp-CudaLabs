
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

using std::cin;
using std::cout;
using std::endl;
using std::setprecision;
using std::scientific;
using std::string;

__global__ void kernel(const double *first, const double *second, unsigned int n, double *results) {
    auto totalThreadsCount = gridDim.x * blockDim.x;
    auto currentThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    for (unsigned int i = currentThreadId; i < n; i += totalThreadsCount) {
        results[i] = first[i] > second[i]
                     ? first[i]
                     : second[i];
    }
}

int main() {
    unsigned int n;
    cin >> n;

    auto *first = new double[n];
    auto *second = new double[n];
    for (int i = 0; i < n; ++i) {
        cin >> first[i];
    }
    for (int i = 0; i < n; ++i) {
        cin >> second[i];
    }

    double *cudaFirst;
    double *cudaSecond;
    hipMalloc(&cudaFirst, sizeof(double) * n);
    hipMalloc(&cudaSecond, sizeof(double) * n);
    hipMemcpy(cudaFirst, first, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(cudaSecond, second, sizeof(double) * n, hipMemcpyHostToDevice);
    delete[] first;
    delete[] second;

    double *cudaResults;
    hipMalloc(&cudaResults, sizeof(double) * n);
    kernel<<<1024, 1024>>>(cudaFirst, cudaSecond, n, cudaResults);
    hipFree(cudaFirst);
    hipFree(cudaSecond);

    auto *results = new double[n];
    hipMemcpy(results, cudaResults, sizeof(double) * n, hipMemcpyDeviceToHost);
    hipFree(cudaResults);

    cout << setprecision(10) << scientific;
    for (unsigned int i = 0; i < n; ++i) {
        cout << results[i] << " ";
    }

    delete[] results;

    return 0;
}
